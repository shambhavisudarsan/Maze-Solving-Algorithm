#include "hip/hip_runtime.h"
#include <bits/stdc++.h>

std::vector<std::vector<int>> initializeVectorOfVectors(int n,int m,int val){
    std::vector<std::vector<int>>vector_of_vectors;
    for(int i=0;i<=n;i++){
        std::vector<int>temp;
        for(int j=0;j<=m;j++){
            temp.push_back(val);
        }
        vector_of_vectors.push_back(temp);
    }
    return vector_of_vectors;
}

bool horizontalBreaking(std::string direction, int i,int j ,int n,int m, std::vector<std::vector<char>> &grid){
    if(direction == "->"){
        if(i+1<=n && j-1>=0){
            if (grid[i+1][j-1] == 'B' && grid[i+1][j] == 'U' && grid[i][j-1] == 'U') return true;
        }
        if(i-1>=1 && j-1>=0){
            if(grid[i-1][j-1] == 'B' && grid[i][j-1] == 'U' && grid[i-1][j] == 'U') return true;
        }
        if(j-1 == m) return true;
        else return false;
    }
    else if(direction == "<-"){
        if(i+1<=n && j+1<=m+1){
            if (grid[i+1][j+1] == 'B' && grid[i+1][j] == 'U' && grid[i][j+1] == 'U') return true;
        }
        if(i-1>=1 && j+1<=m+1){
            if(grid[i-1][j+1] == 'B' && grid[i][j+1] == 'U' && grid[i-1][j] == 'U')  return true;
        }
        if(j == 0) return true;
        else return false;
    }
    else return false;
}

int tableLookup(int p,int ny,int i,int n,int m,std::string direction, std::vector<std::vector<int>>satellite_horizontal, std::vector<std::vector<int>>successor_horizontal, std::vector<std::vector<char>>grid){
    if(successor_horizontal[i][p] != -1){
        if(direction == "->" && satellite_horizontal[i][successor_horizontal[i][p]] <= p){
            return successor_horizontal[i][p];
        }
        if(direction == "<-" && satellite_horizontal[i][successor_horizontal[i][p]] >= p){
            return successor_horizontal[i][p];
        }
    }
    successor_horizontal[i][p] = ny;
    if(ny>=1 && ny<=m){
        if(horizontalBreaking(direction,i,ny,n,m,grid)){
            return ny;
        }
        else if(direction == "->" && ((ny-1>=1 && grid[i][ny] == 'B' && grid[i][ny-1] == 'U') || (ny == 0 && grid[i][ny] == 'B'))){
            return ny;
        }
        else if(direction == "<-" && ((ny+1<=m && grid[i][ny] == 'B' && grid[i][ny+1] == 'U') || (ny == m && grid[i][ny] == 'B'))){
            return ny;
        }
        if(direction == "->"){
            successor_horizontal[i][p] = tableLookup(ny,ny+1,i,n,m,direction,satellite_horizontal,successor_horizontal,grid);
        }
        else{
            successor_horizontal[i][p] = tableLookup(ny,ny-1,i,n,m,direction,satellite_horizontal,successor_horizontal,grid);
        }
    }
    return successor_horizontal[i][p];
}

__global__ std::vector<std::vector<int>> satelliteTable(std::string direction,int n,int m,std::vector<std::vector<char>> &grid, std::vector<std::vector<int>> &satellite_table){

    int idx = threadIdx.x+blockDim.x*blockIdx.x; // create thread x index
    int idy = threadIdx.y+blockDim.y*blockIdx.y; // create thread y index

    int previous_breaking_node = 1;
    if(direction == "->"){
        if(idx<=n){
            previous_breaking_node = 1;
            if(idy<m+2){
                if(horizontalBreaking("->",idx,idy,n,m,grid)){
                    satellite_table[idx][idy] = previous_breaking_node;
                    previous_breaking_node = idy;
                }
                else if(grid[idx][idy] == 'B' && grid[idx][idy-1] == 'U'){
                    satellite_table[idx][idy] = previous_breaking_node;
                }
                else if(grid[idx][idy] == 'U' && grid[idx][idy-1] == 'B'){
                    previous_breaking_node = idy;
                }
            }
        }
    }
    else if(direction == "<-"){
        idx = threadIdx.x+blockDim.x*blockIdx.x; // create thread x index
        idy = threadIdx.y+blockDim.y*blockIdx.y; // create thread y index
        idy = m+2-1-idy;
        if(idx<=n){
            previous_breaking_node = m;
            if(idy<m+2){
                if(horizontalBreaking("<-",idx,idy,n,m,grid)){
                    satellite_table[idx][idy] = previous_breaking_node;
                    previous_breaking_node = idy;
                }
                else if(grid[idx][idy] == 'B' && grid[idx][idy+1] == 'U'){
                    satellite_table[idx][idy] = previous_breaking_node;
                }
                else if(grid[idx][idy] == 'U' && grid[idx][idy+1] == 'B'){
                    previous_breaking_node = idy;
                }
            }
        }
    }   
    return satellite_table;
}

void print(int n,int m,std::vector<std::vector<int>>satellite_horizontal_right,std::vector<std::vector<int>>satellite_horizontal_left,std::vector<std::vector<int>>successor_horizontal_right,std::vector<std::vector<int>>successor_horizontal_left){
    std::cout<<"\nSatellite Horizontal Left"<<'\n';
    for(int i=1;i<=n;i++){
        for(int j=0;j<=m;j++){
            std::cout<<satellite_horizontal_left[i][j]<<" ";
        }
        std::cout<<'\n';
    }
    std::cout<<'\n';
    std::cout<<"Satellite Horizontal Right"<<'\n';
    for(int i=1;i<=n;i++){
        for(int j=1;j<=m+1;j++){
            std::cout<<satellite_horizontal_right[i][j]<<" ";
        }
        std::cout<<'\n';
    }
    std::cout<<'\n';
    std::cout<<"Successor Horizontal Right"<<'\n';
    for(int i=1;i<=n;i++){
        for(int j=1;j<=m;j++){
            std::cout<<successor_horizontal_right[i][j]<<" ";
        }
        std::cout<<'\n';
    }
    std::cout<<'\n';
    std::cout<<"Successor Horizontal Left"<<'\n';
    for(int i=1;i<=n;i++){
        for(int j=1;j<=m;j++){
            std::cout<<successor_horizontal_left[i][j]<<" ";
        }
        std::cout<<'\n';
    }

}

int main(){
    int t,n,m,x,y,b; 

    /* Number of Test Cases */
    std::cin>>t;

    /* For each test case, input n and m */
    while(t--){
        std::cin>>n>>m;

        /* Grid Initialization with U*/
        std::vector<std::vector<char>>grid;

        for(int i=0;i<=n;i++){
            std::vector<char>temp;
            for(int j=0;j<=m;j++){
                temp.push_back('U');
            }
            grid.push_back(temp);
        }

        /* Number of blockages */
        std::cin>>b;

        /* Blockages */
        for(int i=0;i<b;i++){
            std::cin>>x>>y;
            grid[x][y] = 'B';
        }

        std::vector<std::vector<int>>table;

        for(int i=0;i<=n;i++){
            std::vector<int>temp;
            for(int j=0;j<=m+2;j++){
                temp.push_back(-1);
            }
            table.push_back(temp);
        }

        hipMalloc(&matrix, n*m*sizeof(char));
        hipMalloc(&satellite_table, n*(m+2)*sizeof(int));
        // hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
        // hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
        cudaCheckErrors("hipMalloc failure");
        hipMemcpy(grid, matrix, n*m*sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(table, satellite_table, n*m*sizeof(char), hipMemcpyHostToDevice);
        // hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
        cudaCheckErrors("hipMemcpy H2D failure");

        // Launch kernel
        dim3 grid(n, n);  // dim3 variable holds 3 dimensions
        dim3 block((m+2+block.x-1)/block.x, (m+2+block.y-1)/block.y);
        std::vector<std::vector<int>>satellite_horizontal_right = satelliteTable<<<grid, block>>>("->", n, m, matrix, satellite_table);
        std::vector<std::vector<int>>satellite_horizontal_left = initializeVectorOfVectors(n,m,-1);
        std::vector<std::vector<int>>successor_horizontal_right = initializeVectorOfVectors(n,m,-1);
        std::vector<std::vector<int>>successor_horizontal_left =  initializeVectorOfVectors(n,m,-1);
        cudaCheckErrors("kernel launch failure");
        hipMemcpy(satellite_horizontal_right, satellite_table, n*m+2*sizeof(int), hipMemcpyDeviceToHost);
        print(n,m,satellite_horizontal_right,satellite_horizontal_left,successor_horizontal_right,successor_horizontal_left);

        /* Initializing satellite vector*/
        // std::vector<std::vector<int>>satellite_horizontal_right = satelliteTable("->",n,m,&matrix);
        // std::vector<std::vector<int>>satellite_horizontal_left = satelliteTable("<-",n,m,&matrix);
        // std::vector<std::vector<int>>successor_horizontal_right = initializeVectorOfVectors(n,m,-1);
        // std::vector<std::vector<int>>successor_horizontal_left = initializeVectorOfVectors(n,m,-1);

        // for(int i=1;i<=n;i++){
        //     for(int j=0;j<=m;j++){
        //         std::cout<<satellite_horizontal_left[i][j]<<" ";
        //     }
        //     std::cout<<'\n';
        // }
        // std::cout<<'\n';
        // for(int i=1;i<=n;i++){
        //     for(int j=1;j<=m+1;j++){
        //         std::cout<<satellite_horizontal_right[i][j]<<" ";
        //     }
        //     std::cout<<'\n';
        // }


        // for(int i=1;i<=n;i++){
        //     for(int j=1;j<=m;j++){
        //         successor_horizontal_right[i][j] = tableLookup(j,j+1,i,n,m,"->",satellite_horizontal_right,successor_horizontal_right,grid);
        //     }
        // }
        // for(int i=1;i<=n;i++){
        //     for(int j=m;j>=1;j--){
        //         successor_horizontal_left[i][j] = tableLookup(j,j-1,i,n,m,"<-",satellite_horizontal_left,successor_horizontal_left,grid);
        //     }
        // }

        // std::cout<<'\n';
        // for(int i=1;i<=n;i++){
        //     for(int j=1;j<=m;j++){
        //         std::cout<<successor_horizontal_right[i][j]<<" ";
        //     }
        //     std::cout<<'\n';
        // }
        // std::cout<<'\n';
        // for(int i=1;i<=n;i++){
        //     for(int j=1;j<=m;j++){
        //         std::cout<<successor_horizontal_left[i][j]<<" ";
        //     }
        //     std::cout<<'\n';
        // }

        // std::cout<<tableLookup(1,2,1,3,n,m,"->",satellite_horizontal_right,successor_horizontal_right,grid);
    }
}
